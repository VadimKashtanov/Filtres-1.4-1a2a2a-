#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

/*
Utilisation :
	./prog2__resultats mdl.bin
*/

void __gain(Mdl_t * mdl, uint t0) {
	uint t1 = t0 + 14*24;	//336h % 16 == 0
	printf("gain t0=%i t1=%i [2 semaines] = %f$\n", t0, t1, mdl_gain(mdl, t0, t1, 3));
};

void __pred(Mdl_t * mdl, uint t0, uint t1) {
	printf("pred %i a %i = %f%%\n", t0, t1, 100*mdl_pred(mdl, t0, t1, 3));
}

int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");  charger_tout();
	//
	if (argc == 2) {
		Mdl_t * mdl = ouvrire_mdl(argv[1]);
		mdl_verif(mdl);
		//
		uint t0 = DEPART;
		uint t1 = ROND_MODULO(FIN, 16);
		//
		__pred(mdl, t0, t1);
		__pred(mdl, t0, t0+16*1000);
		__pred(mdl, t0+16*1000, t1-16*1000);
		__pred(mdl, t1-16*1000, t1);
		__pred(mdl, t1-16*100, t1);
		//
		titre("Gain en 2 semaines");
		__gain(mdl, t0);
		__gain(mdl, t0+5000);
		__gain(mdl, t0+10000);
		__gain(mdl, t1-2*365*24);
		__gain(mdl, t1-356*24);
		__gain(mdl, t1-3*30*24);
		__gain(mdl, t1-14*24);
		//
		liberer_mdl(mdl);
	} else {
		ERR("./prog2__resultats mdl.bin")
	}
	liberer_tout();
}