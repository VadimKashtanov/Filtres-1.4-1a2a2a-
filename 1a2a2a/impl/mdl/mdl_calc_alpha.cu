#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_moy_dp(Mdl_t * mdl, uint c) {
	uint POIDS = (c==0 ? mdl->ST[0]*N : mdl->ST[c]*(mdl->ST[c-1]+1));
	//
	float * dp = gpu_vers_cpu<float>(
		(c==0 ? mdl->df : mdl->dp__d[c]),
		POIDS);
	float moy = 0;
	FOR(0, i, POIDS) {
		moy += fabs(dp[i]);
	};
	free(dp);
	return moy / (float)POIDS;
};