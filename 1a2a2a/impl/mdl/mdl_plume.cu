#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void plumer_mdl(Mdl_t * mdl) {
	printf("\033[4m Plume mdl\033[0m\n");
	printf("Filtres : %i\n", mdl->ST[0]);
	printf("Lignes des bloques : ");
	FOR(0, i, mdl->bloques) printf("%i ", mdl->lignes[i]);
	printf("\n");
	uint POIDS = 0;
	FOR(1, c, C) {
		POIDS += poids_couche(mdl, c);
		printf("%2.i | Y=%i (P=%i)\n", c, mdl->ST[c], poids_couche(mdl, c));
	}
	printf("Quantitée poids = %i\n", POIDS);
	printf(" --- fin plume mdl ---\n");
};

void comportement(Mdl_t * mdl, uint t0, uint t1) {
	mdl_f(mdl, t0, t1, 3);
	mdl_gpu_vers_cpu(mdl);
	FOR(0, c, C) {
		printf("###=== c=%i ===###\n", c);
		FOR(0, i, mdl->ST[c]) {
			printf("%3.i| ", i);
			FOR(t0, t, t1) printf("%+f ", mdl->y[c][i+t*mdl->ST[c]]);
			printf("\n");
		}
	}
};

void comportement_c1(Mdl_t * mdl, uint t0, uint t1) {
	mdl_f(mdl, t0, t1, 3);
	mdl_gpu_vers_cpu(mdl);
	uint c = C-1;
	printf("###=== c=%i ===###\n", c);
	FOR(0, i, mdl->ST[c]) {
		printf("%3.i| ", i);
		FOR(t0, t, t1) printf("%+f ", mdl->y[c][i+t*mdl->ST[c]]);
		printf("\n");
	};
};

void cmp_dy_dp(Mdl_t * mdl, uint t0, uint t1) {
	printf(" ########## COMPARER DY #########\n");
	FOR(0, c, C) {
		float * m = gpu_vers_cpu<float>(mdl->dy__d[c], mdl->ST[c]*t1);
		printf("###=== c=%i ===###\n", c);
		FOR(0, i, mdl->ST[c]) {
			printf("%3.i| cpu : ", i);
			FOR(t0, t, t1) printf("%+f ", mdl->dy[c][i+t*mdl->ST[c]]);

			printf(" gpu : ");

			FOR(t0, t, t1) printf("%+f ", m[i+t*mdl->ST[c]]);
			printf("\n");
		}
		free(m);
	}

	printf(" ########## COMPARER DP #########\n");
	FOR(1, c, C) {
		float * m = gpu_vers_cpu<float>(mdl->dp__d[c], (mdl->ST[c-1]+1)*mdl->ST[c]);
		printf("###=== c=%i ===###\n", c);
		FOR(0, i, (mdl->ST[c-1]+1)*mdl->ST[c]) {
			printf("%3.i| cpu : ", i);
			printf("%+f ", mdl->dp[c][i]);

			printf(" gpu : ");

			printf("%+f ", m[i]);
			printf("\n");
		}
		free(m);
	}
};

void mdl_plume_grad(Mdl_t * mdl, uint t0, uint t1) {
	mdl_aller_retour(mdl, t0, t1, 3);
	//mdl_gpu_vers_cpu(mdl);
	//
	printf(" Couche 0 \033[93mX=%i Y=%i\033[0m", mdl->ST[0], mdl->ST[0]);
	uint FILTRES = mdl->ST[0]*N;
	float * _grad = gpu_vers_cpu<float>(mdl->df__d, FILTRES);
	float moyenne = 0;
	float min=_grad[0], max=_grad[0];
	FOR(0, i, FILTRES) {
		if (_grad[i] > max) max = _grad[i];
		if (_grad[i] < min) min = _grad[i];
		moyenne += fabs(_grad[i]);
	}
	printf("[Grad:Min=%+f;Max=%+f;Moyenne=(-/+)%f]\n",
		min, max, moyenne / FILTRES
	);
	free(_grad);
	//
	//
	FOR(1, c, C) {
		printf(" Couche %i \033[93mX=%i Y=%i\033[0m", c, mdl->ST[c-1], mdl->ST[c]);
		uint POIDS = (mdl->ST[c-1]+1)*mdl->ST[c];
		float * _grad = gpu_vers_cpu<float>(mdl->dp__d[c], POIDS);
		float * ____p = gpu_vers_cpu<float>(mdl->p__d[c], POIDS);
		//
		//	Grad
		//
		float moyenne = 0;
		float min=_grad[0], max=_grad[0];
		FOR(0, i, POIDS) {
			if (_grad[i] > max) max = _grad[i];
			if (_grad[i] < min) min = _grad[i];
			moyenne += fabs(_grad[i]);
		}
		printf("[Grad:Min=%+f;Max=%+f;Moyenne=(-/+)%f]",
			min, max, moyenne / POIDS
		);
		free(_grad);
		//
		//	P
		//
		moyenne = 0;
		min=____p[0], max=____p[0];
		FOR(0, i, POIDS) {
			if (____p[i] > max) max = ____p[i];
			if (____p[i] < min) min = ____p[i];
			moyenne += fabs(____p[i]);
		}
		printf("[P:Min=%+f;Max=%+f;Moyenne=(-/+)%f]\n",
			min, max, moyenne / POIDS
		);
		free(____p);
	}
};