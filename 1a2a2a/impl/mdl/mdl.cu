#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

uint poids_couche(Mdl_t * mdl, uint c) {
	return mdl->ST[c] * (mdl->ST[c-1]+1);
}

uint entrees_couche(Mdl_t * mdl, uint c) {
	return mdl->ST[c-1];
}

PAS_OPTIMISER()
Mdl_t * cree_mdl(uint * ST, uint * activation, uint bloques, uint f_par_bloque, uint * lignes) {
	ASSERT(ST[C-1] == 1);
	
	Mdl_t * mdl = alloc<Mdl_t>(1);

	mdl->activation = copier<uint>(activation, C);

	mdl->ST = copier<uint>(ST, C);
	mdl->ST__d = cpu_vers_gpu<uint>(ST, C);

	mdl->bloques = bloques;
	mdl->f_par_bloque = f_par_bloque;

	mdl->lignes = copier<uint>(lignes, bloques);
	mdl->lignes__d = cpu_vers_gpu<uint>(lignes, bloques);

	ASSERT(ST[0] == bloques * f_par_bloque);

	//	Espace ram intel
	mdl->f = lst_rnd(ST[0] * N, -1, 1);
	mdl->f_locd_s = alloc<float>(ST[0]*PRIXS);
	mdl->f_locd_d = alloc<float>(ST[0]*PRIXS);
	mdl->dif_f = alloc<float>(ST[0]*(N-1));
	FOR(0, i, ST[0]) {
		//	Trouver min, max
		float min = mdl->f[i*N + 0];
		float max = mdl->f[i*N + 0];
		FOR(1, j, N) {
			if (mdl->f[i*N + j] < min) min = mdl->f[i*N + j];
			if (mdl->f[i*N + j] > max) max = mdl->f[i*N + j];
		}

		//	Normaliser (e-min)/(max-min)
		FOR(0, j, N) mdl->f[i*N + j] = (mdl->f[i*N + j]-min)/(max-min);

		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N + j+1]-mdl->f[i*N + j];
		}
	}

	mdl->f__d = cpu_vers_gpu<float>(mdl->f, ST[0] * N);
	mdl->f_locd_s__d = cudalloc<float>(ST[0]*PRIXS);
	mdl->f_locd_d__d = cudalloc<float>(ST[0]*PRIXS);
	mdl->dif_f__d = cpu_vers_gpu<float>(mdl->dif_f, ST[0] * (N-1));

	mdl->df = alloc<float>(ST[0] * N);
	mdl->df__d = cudalloc<float>(ST[0] * N);

	{	//c = 1
		uint c = 0;
		mdl->p[c] = alloc<float>(0);
		mdl->y[c] = alloc<float>(ST[c]*PRIXS);
		mdl->locd[c] = alloc<float>(0);
		mdl->dy[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dp[c] = alloc<float>(0);
		//
		mdl->p__d[c] = alloc<float>(0);
		mdl->y__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->locd__d[c] = cudalloc<float>(0);
		mdl->dy__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dp__d[c] = alloc<float>(0);
	}

	FOR(1, c, C) {
		float X = (float)ST[c-1];
		float A = -0.05 - 0.7*fabs(1-(X-mdl->ST[C-2])/(mdl->ST[0]-mdl->ST[C-2]));
		float B = +0.05 + 0.7*fabs(1-(X-mdl->ST[C-2])/(mdl->ST[0]-mdl->ST[C-2]));
		mdl->p[c] = lst_rnd((ST[c-1]+1)*ST[c], A, B);
		mdl->y[c] = alloc<float>(ST[c]*PRIXS);
		mdl->locd[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dy[c] = alloc<float>(ST[c]*PRIXS);
		mdl->dp[c] = alloc<float>((ST[c-1]+1)*ST[c]);

		mdl->p__d[c] = cpu_vers_gpu(mdl->p[c], (ST[c-1]+1)*ST[c]);
		mdl->y__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->locd__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dy__d[c] = cudalloc<float>(ST[c]*PRIXS);
		mdl->dp__d[c] = cudalloc<float>((ST[c-1]+1)*ST[c]);
	}

	mdl_diff_f(mdl);

	return mdl;
};

PAS_OPTIMISER()
void mdl_verif(Mdl_t * mdl) {
	float * r = gpu_vers_cpu<float>(mdl->dif_f__d, mdl->ST[0]*(N-1));
	FOR(0, i, (mdl->ST[0]*(N-1))) ASSERT(fabs(r[i]-mdl->dif_f[i]) < 0.01);
	free(r);
	//
	FOR(1, c, C) {
		float * r = gpu_vers_cpu<float>(mdl->p__d[c], (mdl->ST[c-1]+1)*mdl->ST[c]);
		FOR(0, i, (mdl->ST[c-1]+1)*mdl->ST[c]) ASSERT(fabs(r[i]-mdl->p[c][i]) < 0.01);
		free(r);
	}
};

PAS_OPTIMISER()
void mdl_diff_f(Mdl_t * mdl) {
	FOR(0, i, mdl->ST[0]) {
		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N + j+1]-mdl->f[i*N + j];
		}
	}
	CONTROLE_CUDA(hipMemcpy(mdl->dif_f__d, mdl->dif_f, sizeof(float)*mdl->ST[0]*(N-1), hipMemcpyHostToDevice));
};

PAS_OPTIMISER()
void mdl_gpu_vers_cpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->f,     mdl->f__d,     sizeof(float)*mdl->ST[0]*N, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->f_locd_s,     mdl->f_locd_s__d,     sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->f_locd_d,     mdl->f_locd_d__d,     sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->df,     mdl->df__d,     sizeof(float)*mdl->ST[0]*N, hipMemcpyDeviceToHost));
	//
	mdl_diff_f(mdl);
	//
	CONTROLE_CUDA(hipMemcpy(mdl->y[0],  mdl->y__d[0],  sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->dy[0], mdl->dy__d[0], sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyDeviceToHost));
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p[c],    mdl->p__d[c],    sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->y[c],    mdl->y__d[c],    sizeof(float)*mdl->ST[c]*PRIXS, 		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->locd[c], mdl->locd__d[c], sizeof(float)*mdl->ST[c]*PRIXS,  		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dy[c],   mdl->dy__d[c],   sizeof(float)*mdl->ST[c]*PRIXS,  		   hipMemcpyDeviceToHost));
		CONTROLE_CUDA(hipMemcpy(mdl->dp[c],   mdl->dp__d[c],   sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyDeviceToHost));
	}
}

PAS_OPTIMISER()
void mdl_cpu_vers_gpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->f__d,     mdl->f,     sizeof(float)*mdl->ST[0]*N, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->dif_f__d,     mdl->dif_f,     sizeof(float)*mdl->ST[0]*(N-1), hipMemcpyHostToDevice));
	//
	CONTROLE_CUDA(hipMemcpy(mdl->f_locd_s__d,     mdl->f_locd_s,     sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->f_locd_d__d,     mdl->f_locd_d,     sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->df__d,     mdl->df,     sizeof(float)*mdl->ST[0]*N, hipMemcpyHostToDevice));
	//
	CONTROLE_CUDA(hipMemcpy(mdl->y__d[0],  mdl->y[0],  sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->dy__d[0], mdl->dy[0], sizeof(float)*mdl->ST[0]*PRIXS, hipMemcpyHostToDevice));
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemcpy(mdl->p__d[c],    mdl->p[c],    sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->y__d[c],    mdl->y[c],    sizeof(float)*mdl->ST[c]*PRIXS, 				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->locd__d[c], mdl->locd[c], sizeof(float)*mdl->ST[c]*PRIXS,  				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dy__d[c],   mdl->dy[c],   sizeof(float)*mdl->ST[c]*PRIXS,  				   hipMemcpyHostToDevice));
		CONTROLE_CUDA(hipMemcpy(mdl->dp__d[c],   mdl->dp[c],   sizeof(float)*(mdl->ST[c-1]+1)*mdl->ST[c], hipMemcpyHostToDevice));
	}
};

PAS_OPTIMISER()
void liberer_mdl(Mdl_t * mdl) {
	CONTROLE_CUDA(hipFree(mdl->ST__d));
	CONTROLE_CUDA(hipFree(mdl->lignes__d));
	CONTROLE_CUDA(hipFree(mdl->f__d));
	CONTROLE_CUDA(hipFree(mdl->dif_f__d));
	//
	CONTROLE_CUDA(hipFree(mdl->f_locd_s__d));
	CONTROLE_CUDA(hipFree(mdl->f_locd_d__d));
	CONTROLE_CUDA(hipFree(mdl->df__d));
	//
	free(mdl->lignes);
	free(mdl->ST);
	free(mdl->activation);
	free(mdl->f);
	free(mdl->dif_f);
	//
	free(mdl->f_locd_d);
	free(mdl->f_locd_s);
	free(mdl->df);
	//
	{
		uint c = 0;
		free(mdl->y[c]);
		free(mdl->dy[c]);

		CONTROLE_CUDA(hipFree(mdl->y__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
	}

	FOR(1, c, C) {
		free(mdl->p[c]);
		free(mdl->y[c]);
		free(mdl->locd[c]);
		free(mdl->dy[c]);
		free(mdl->dp[c]);

		CONTROLE_CUDA(hipFree(mdl->p__d[c]));
		CONTROLE_CUDA(hipFree(mdl->y__d[c]));
		CONTROLE_CUDA(hipFree(mdl->locd__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dy__d[c]));
		CONTROLE_CUDA(hipFree(mdl->dp__d[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_cpu(Mdl_t * mdl) {
	memset(mdl->df, 0, sizeof(float)*mdl->ST[0]*N);
	memset(mdl->dy[0], 0, sizeof(float)*mdl->ST[0]*PRIXS);
	//
	FOR(1, c, C) {
		memset(mdl->dy[c], 0, sizeof(float)*mdl->ST[c]*PRIXS);
		memset(mdl->dp[c], 0, sizeof(float)*((mdl->ST[c-1]+1)*mdl->ST[c]));
	}
};

PAS_OPTIMISER()
void mdl_zero_deriv_gpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemset(mdl->df__d, 0, sizeof(float)*mdl->ST[0]*N));
	//
	CONTROLE_CUDA(hipMemset(mdl->dy__d[0], 0, sizeof(float)*mdl->ST[0]*PRIXS));
	//
	FOR(1, c, C) {
		CONTROLE_CUDA(hipMemset(mdl->dy__d[c], 0, sizeof(float)*mdl->ST[c]*PRIXS));
		CONTROLE_CUDA(hipMemset(mdl->dp__d[c], 0, sizeof(float)*((mdl->ST[c-1]+1)*mdl->ST[c])));
	}
};