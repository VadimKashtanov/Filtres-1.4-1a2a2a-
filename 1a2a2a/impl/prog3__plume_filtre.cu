#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

/*
Utilisation :
	./prog0__plume_filtre mdl.bin bloque f_dans_bloque
*/

int main(int argc, char ** argv) {
	srand(0);
	hipSetDevice(0);
	titre(" Charger tout ");  charger_tout();
	//
	if (argc == 4) {
		Mdl_t * mdl = ouvrire_mdl(argv[1]);
		//
		char cmd[1000];
		//
		uint depart = N*atoi(argv[2])*mdl->f_par_bloque + atoi(argv[3])*N;
		snprintf(cmd, 1000, "python3 -c \"import matplotlib.pyplot as plt;plt.plot([%f,%f,%f,%f,%f,%f,%f,%f]);plt.show()\"",
			mdl->f[depart + 0],
			mdl->f[depart + 1],
			mdl->f[depart + 2],
			mdl->f[depart + 3],
			mdl->f[depart + 4],
			mdl->f[depart + 5],
			mdl->f[depart + 6],
			mdl->f[depart + 7]
		);
		//
		SYSTEM(cmd);
		liberer_mdl(mdl);
		//
	} else {
		ERR("./prog0__plume_filtre mdl.bin bloque f_dans_bloque")
	}
	liberer_tout();
};