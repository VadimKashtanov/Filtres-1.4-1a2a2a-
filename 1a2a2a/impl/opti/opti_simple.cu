#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static __global__ void kerd_opti_simple(
	float * p, float * dp, float alpha, uint P, float div)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		p[thx] -= alpha * dp[thx] / div;
	}
};

static __global__ void kerd_opti_simple_masque(
	float * p, float * dp, float alpha, uint P, float div, uint * masque)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		if (masque[thx] == NON_MASQUEE)
			p[thx] -= alpha * dp[thx] / div;
	}
};

void opti_simple(Mdl_t * mdl, float * alpha, float div, uint ** masque) {
	//	Filtres
	uint FILTRES = mdl->ST[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
	if (masque == 0) {
		kerd_opti_simple<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->f__d, mdl->df__d, alpha[0], FILTRES, div);
	} else {
		kerd_opti_simple_masque<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->f__d, mdl->df__d, alpha[0], FILTRES, div, masque[0]
		);
	}
	//	Poids
	FOR(1, c, C) {
		uint P = (mdl->ST[c-1]+1)*mdl->ST[c];
		if (masque == 0) {
			kerd_opti_simple<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], P, div
			);
		} else {
			kerd_opti_simple_masque<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], alpha[c], P, div, masque[c]
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_gpu_vers_cpu(mdl);
	mdl_diff_f(mdl);
};