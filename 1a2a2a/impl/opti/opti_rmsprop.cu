#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define rms_alpha 0.9

static __global__ void kerd_opti_rmsprop(
	float * p, float * dp, float * g,
	float alpha, uint P, float div)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		float _grad = dp[thx] / div;
		float _g = rms_alpha*g[thx] + _grad*_grad;
		p[thx] -= alpha * _grad / (sqrtf(_g) + 1e-5);
		g[thx] = _g;
	}
};

static __global__ void kerd_opti_rmsprop_masque(
	float * p, float * dp, float * g,
	float alpha, uint P, float div, uint * masque)
{
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;

	if (thx < P) {
		if (masque[thx] == NON_MASQUEE) {
			float _grad = dp[thx] / div;
			float _g = rms_alpha*g[thx] + _grad*_grad;
			p[thx] -= alpha * _grad / (sqrtf(_g) + 1e-5);
			g[thx] = _g;
		}
	}
};

Rmsprop_t * cree_rmsprop(
	Mdl_t * mdl)
{
	Rmsprop_t * ret = alloc<Rmsprop_t>(1);
	ret->g[0] = cudazero<float>(mdl->ST[0]*N);
	FOR(1, c, C) ret->g[c] = cudazero<float>(poids_couche(mdl, c));
	return ret;
};

void liberer_rmsprop(Rmsprop_t * rmsprop) {
	FOR(0, c, C) cudafree<float>(rmsprop->g[c]);
	free(rmsprop);
};

void opti_rmsprop(
	Mdl_t * mdl, Rmsprop_t * rmsprop,
	float * alpha, float div, uint ** masque)
{
	//	Filtres
	uint FILTRES = mdl->ST[0];	//pas de *N, car c'est le filtre qu'on ignore, pas les points
	if (masque == 0) {
		kerd_opti_rmsprop<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->f__d, mdl->df__d, rmsprop->g[0], alpha[0], FILTRES, div);
	} else {
		kerd_opti_rmsprop_masque<<<dim3(KERD(FILTRES, 256)), dim3(256)>>>(
			mdl->f__d, mdl->df__d, rmsprop->g[0], alpha[0], FILTRES, div, masque[0]
		);
	}
	//	Poids
	FOR(1, c, C) {
		uint P = (mdl->ST[c-1]+1)*mdl->ST[c];
		
		if (masque == 0) {
			kerd_opti_rmsprop<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], P, div
			);
		} else {
			kerd_opti_rmsprop_masque<<<dim3(KERD(P, 1024)), dim3(1024)>>>(
				mdl->p__d[c], mdl->dp__d[c], rmsprop->g[c],
				alpha[c], P, div, masque[c]
			);
		}
	};
	ATTENDRE_CUDA();
	mdl_gpu_vers_cpu(mdl);
	mdl_diff_f(mdl);
};