#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../../impl_tmpl/tmpl_etc.cu"

//	===============================================================

/*static __global__ void kerd_nvidia_score_powf(
	float * y, uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		y[depart+_t] = SCORE(
			y[depart+_t], _PRIXS[(depart+_t)+L], _PRIXS[(depart+_t)]
		);
	}
};*/

static __global__ void kerd_nvidia_score_somme(
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	float s = 0;
	FOR(0, i, T) {
		s += cuda_SCORE(
			y[depart+i], _PRIXS[(depart+i)+L], _PRIXS[(depart+i)]
		);//y[depart+i];
	}
	*score = s / (float)T*100;
};

float nvidia_score(float * y, uint depart, uint T)
{
	float * score__d = cudalloc<float>(1);
	/*kerd_nvidia_score_powf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		y, depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();*/
	kerd_nvidia_score_somme<<<1,1>>>(
		y, depart, T,
		score__d, prixs__d
	);
	ATTENDRE_CUDA();
	float _score;
	CONTROLE_CUDA(hipMemcpy(&_score, score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(score__d));
	return _score;
};

//	===============================================================

static __global__ void kerd_nvidia_prediction_somme(
	float * y, uint depart, uint T,
	float * pred, float * _PRIXS)
{
	/*
	float _p = 0;
	FOR(0, i, T) {
		if (cuda_signe((y[depart+i])) == cuda_signe((_PRIXS[depart+i+L]/_PRIXS[depart+i]-1)))
			_p += 1.0;
	}
	pred[0] = (_p / (float)T);*/
	uint thx = threadIdx.x + blockIdx.x * blockDim.x;
	if (thx < T) {
		float p1 = _PRIXS[depart+thx+L];
		float p0 = _PRIXS[depart+thx];
		atomicAdd(
			pred,
			1.0*(uint)(cuda_signe((y[depart+thx])) == cuda_signe((p1/p0-1)))
		);
	};
};

float nvidia_prediction(float * y, uint depart, uint T) {
	float * pred__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(pred__d, 0, 1*sizeof(float)));
	kerd_nvidia_prediction_somme<<<dim3(KERD(T,1024)),dim3(1024)>>>(
		y, depart, T,
		pred__d, prixs__d
	);
	ATTENDRE_CUDA();
	float _pred;
	CONTROLE_CUDA(hipMemcpy(&_pred, pred__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	cudafree<float>(pred__d);
	return _pred / (float)T;
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		dy[depart+_t] = cuda_dSCORE(
			y[depart+_t], _PRIXS[(depart+_t)+L], _PRIXS[(depart+_t)]
		) / (float)T*100;
	}
};

void d_nvidia_score(float * y, float * dy, uint depart, uint T) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		y, dy,
		depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();
};