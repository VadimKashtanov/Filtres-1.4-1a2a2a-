#include "hip/hip_runtime.h"
#include "dot1d.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

void intel_dot1d(
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd)
{
#pragma omp parallel
#pragma omp for
	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float s = p[_y*(X+1)+(X+1-1)];
			FOR(0, k, X) {
				float __x = x[(depart+t)*X+k];
				float __p = p[_y*(X+1)+k];
				s += __x * __p;
			}
			float a = ACTIV(s);
			y[(depart+t)*Y+_y]    = a;
			locd[(depart+t)*Y+_y] = dACTIV(s, a);
		}
	}
}

void d_intel_dot1d(
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
//#pragma omp parallel
//#pragma omp for
/*	FOR(0, t, T) {
		FOR(0, _y, Y) {
			float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			dp[_y*(X+1)+(X+1-1)] += _locd;
			FOR(0, k, X) {
				//s += x[t*X+k] * p[y*(X+1)+k];
				dx[(depart+t)*X+k]      += _locd * p[_y*(X+1)+k];
				dp[_y*(X+1)+k] += _locd * x[(depart+t)*X+k];
			}
		}
	}*/

	//dx = (p @ ((y-_y)*dtanh(x@p)).T).T
#pragma omp parallel
#pragma omp for
	FOR(0, t, T) {
		FOR(0, _x, X) {
			//float _locd = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];
			float s = 0;
			FOR(0, k, Y) {
				float __x = p[k*(X+1)+_x];//x[(depart+t)*X+k];
				float __p = locd[(depart+t)*Y+k] * dy[(depart+t)*Y+k];//p[_y*(X+1)+k];
				s += __x * __p;
			}
			dx[(depart+t)*X+_x]   = s;
		}
	}

	//dp = x.T @ ((y-_y)*dtanh(x@p))
#pragma omp parallel
#pragma omp for
	FOR(0, _y, Y) {
		float dbiais = 0;
		FOR(0, _x, X) {
			float s = 0;
			FOR(0, t, T) {
				float __x = locd[(depart+t)*Y+_y] * dy[(depart+t)*Y+_y];//x[(depart+t)*X+k];
				float __p = x[(depart+t)*X+_x];//p[_y*(X+1)+k];
				s += __x * __p;
				if (_x == 0) {	//	Biais
					dbiais += __x;
				}
			}
			dp[_y*(X+1)+_x] = s;
		}
		dp[_y*(X+1)+(X+1-1)] = dbiais;
	}
}

//	=========================================================

void nvidia_dot1d(
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd,
	uint mode)
{
	if (mode == 0) {
		nvidia_dot1d_naive(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd);
	} else if (mode == 1) {
		nvidia_dot1d_shared(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd);
	} else if (mode == 2) {
		nvidia_dot1d_shared_2_16(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd);
	} else {
		ERR("Pas de mode %i pour cuda f(x)", mode);
	}
}

//	----------------------------

void d_nvidia_dot1d(
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp,
	uint mode)
{
	if (mode == 0) {
		d_nvidia_dot1d_naive(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	} else if (mode == 1) {
		d_nvidia_dot1d_shared(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	} else if (mode == 2) {
		d_nvidia_dot1d_shared_2_16(	//	2 versions : 1x stricte et 1x non stricte
			X, Y,
			depart, T,
			x, y,
			p,
			locd,
			dy,
			dx,
			dp);
	} else {
		ERR("Pas de mode %i pour cuda f'(x)", mode);
	}
}

//	----------------------------

static void verif_intel_papier() {	//verifier f(x) avec les valeurs a la main
	uint depart = 0;
	uint X = 3, T = 2, Y = 4; 
	float x[] = {
		//	t = 0
		.1,.2,.3,
		//	t = 1
		.4,.5,.6
	};
	float p[] = {
		//	y = 0
		.1,.2,.3, .4,
		//	y = 1
		.5,.6,.7, .8,
		//	y = 2
		.9,.10,.11, .12,
		//	y = 3
		.13,.14,.15, .16
	};
	float s[] = {
		//	## t = 0 ##
		.1*.1 + .2*.2 + .3*.3 + .4,
		.1*.5 + .2*.6 + .3*.7 + .8,
		.1*.9 + .2*.10 + .3*.11 + .12,
		.1*.13 + .2*.14 + .3*.15 + .16,
		//	## t = 1 ##
		.4*.1 + .5*.2 + .6*.3 + .4,
		.4*.5 + .5*.6 + .6*.7 + .8,
		.4*.9 + .5*.10 + .6*.11 + .12,
		.4*.13 + .5*.14 + .6*.15 + .16
	};
	float y[] = {
		//	## t = 0 ##
		(float)ACTIV(s[0]),
		(float)ACTIV(s[1]),
		(float)ACTIV(s[2]),
		(float)ACTIV(s[3]),
		//	## t = 1 ##
		(float)ACTIV(s[4]),
		(float)ACTIV(s[5]),
		(float)ACTIV(s[6]),
		(float)ACTIV(s[7]),
	};
	float locd[] = {
		dACTIV(s[0], y[0]),
		dACTIV(s[1], y[1]),
		dACTIV(s[2], y[2]),
		dACTIV(s[3], y[3]),

		dACTIV(s[4], y[4]),
		dACTIV(s[5], y[5]),
		dACTIV(s[6], y[6]),
		dACTIV(s[7], y[7])
	};

	float _y[Y*T];
	float _l[Y*T];

	intel_dot1d(
		X, Y,
		depart, T,
		x, _y, p, _l);

	titre("intel dot1d papier");
	puts("paper y - intel y\n");
	comparer_lst(y, _y, Y*T, 0.001);
	puts("paper locd - intel locd\n");
	comparer_lst(locd, _l, Y*T, 0.001);
};

static void verif_intel_1e5() {	//verifier f(x) avec les valeurs a la main
	uint depart = 0;
	uint X = 4, T = 3, Y = 7;
	float * x = lst_rnd(X*T, -1, 1);
	float * y = alloc<float>(T*Y);
	float * p = lst_rnd((X+1)*Y, -1, 1);
	float * l = alloc<float>(T*Y);

	float * sorties = lst_rnd(Y*T, -1, 1);

	float * dp = zero<float>((X+1)*Y);
	float * dx = zero<float>(T*X);
	float * dy = zero<float>(T*Y);

	intel_dot1d(
		X, Y,
		depart, T,
		x, y, p, l);

	//	S
	float S = 0;
	FOR(0, i, Y*T) S += powf((y[i]-sorties[i]), 2) / 2;// / (Y*T);

	//	dS
	FOR(0, i, Y*T) dy[i] = (y[i]-sorties[i]);// / (Y*T);

	d_intel_dot1d(
		X, Y,
		depart, T,
		x, y, p, l,
		dy, dx, dp);

	float * _dp = zero<float>((X+1)*Y);
	float * _dx = zero<float>(T*X);

	float _1E5 = 1e-3;

	//	_dp
	float Sx;
	FOR(0, i, X*Y) {
		p[i] += _1E5;
		//
		intel_dot1d(
			X, Y,
			depart, T,
			x, y, p, l);
		p[i] -= _1E5;
		//
		Sx = 0;
		FOR(0, j, Y*T) Sx += powf((y[j]-sorties[j]), 2) / 2;// / (Y*T);
		//
		_dp[i] = (Sx-S)/_1E5;
	};

	//	_dx
	FOR(0, i, X*T) {
		x[i] += _1E5;
		//
		intel_dot1d(
			X, Y,
			depart, T,
			x, y, p, l);
		x[i] -= _1E5;
		//
		Sx = 0;
		FOR(0, j, Y*T) Sx += powf((y[j]-sorties[j]), 2) / 2;// / (Y*T);
		//
		_dx[i] = (Sx-S)/_1E5;
	};

	titre("intel dot1d 1e-5");
	puts("1e5 dp - intel dp\n");
	comparer_lst(_dp, dp, Y*X, 0.001);
	puts("1e5 dx - intel dx\n");
	comparer_lst(_dx, dx, X*T, 0.001);

	free(x);
	free(y);
	free(p);
	free(l);
	free(sorties);
	free(dp);
	free(dx);
	free(dy);
	free(_dp);
	free(_dx);
};

static void verif_nvidia_intel()
{
	uint depart = 0;
	uint X = 3*16, T = 2*16, Y = 4*16;
	//
	float * x = lst_rnd(X*T, -1, 1);
	float * y = alloc<float>(T*Y);
	float * p = lst_rnd((X+1)*Y, -1, 1);
	float * l = alloc<float>(T*Y);
	float * sorties = lst_rnd(Y*T, -1, 1);
	float * dp = zero<float>((X+1)*Y);
	float * dx = zero<float>(T*X);
	float * dy = zero<float>(T*Y);

	intel_dot1d(
		X, Y,
		depart, T,
		x, y,
		p,
		l);
	//	dS
	FOR(0, i, Y*T) dy[i] = (y[i]-sorties[i]) / (Y*T);
	//
	d_intel_dot1d(
		X, Y,
		depart, T,
		x, y, p, l,
		dy, dx, dp);

	float * x__d = cpu_vers_gpu<float>(x, X*T);
	float * y__d = cpu_vers_gpu<float>(y, Y*T);
	float * p__d = cpu_vers_gpu<float>(p, (X+1)*Y);
	float * l__d = cpu_vers_gpu<float>(l, Y*T);
	float * sorties__d = cpu_vers_gpu<float>(sorties, Y*T);
	float * dp__d = cpu_vers_gpu<float>(dp, (X+1)*Y);
	float * dx__d = cpu_vers_gpu<float>(dx, X*T);
	float * dy__d = cpu_vers_gpu<float>(dy, T*Y);
	//
	FOR(0, mode, 3) {
		CONTROLE_CUDA(hipMemset(dp__d, 0, sizeof(float) * (X+1)*Y));
		CONTROLE_CUDA(hipMemset(dx__d, 0, sizeof(float) * X*T));
		CONTROLE_CUDA(hipMemset(y__d, 0, sizeof(float) * T*Y));
		CONTROLE_CUDA(hipMemset(l__d, 0, sizeof(float) * Y*T));
		//
		nvidia_dot1d(
			X, Y,
			depart, T,
			x__d, y__d,
			p__d,
			l__d,
			mode);
		d_nvidia_dot1d(
			X, Y,
			depart, T,
			x__d, y__d,
			p__d,
			l__d,
			dy__d,
			dx__d,
			dp__d,
			mode);
		printf("########## Verif dot1d MODE %i #######\n", mode);

		puts("y__d\n");
		comparer_lst(y, gpu_vers_cpu<float>(y__d, (T)*Y), T*Y, 0.001);
		puts("l__d\n");
		comparer_lst(l, gpu_vers_cpu<float>(l__d, T*Y), T*Y, 0.001);
		puts("dp__d\n");
		comparer_lst(dp, gpu_vers_cpu<float>(dp__d, (X+1)*Y), (X+1)*Y, 0.001);
		puts("dx__d\n");
		comparer_lst(dx, gpu_vers_cpu<float>(dx__d, X*T), X*T, 0.001);
	}
};

void verif_do1d() {
	verif_intel_papier();
	verif_intel_1e5();
	verif_nvidia_intel();
};