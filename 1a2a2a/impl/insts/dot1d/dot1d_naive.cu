#include "hip/hip_runtime.h"
#include "dot1d.cuh"

#define BLOQUE_T 32
#define BLOQUE_Y 32

static __global__ void kerd(	//	2 version : 1 stricte et une non stricte
	uint ACTIVATION,
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float s = p[_y*(X+1) + (X+1-1)];
		FOR(0, i, X) s += x[(depart+_t)*X + i] * p[_y*(X+1) + i];
		float a = ACTIV(ACTIVATION, s);
		y[(depart+_t)*Y + _y] = a;
		locd[(depart+_t)*Y + _y] = dACTIV(ACTIVATION, s,a);
	}
};

void nvidia_dot1d_naive(	//	2 version : 1 stricte et une non stricte
	uint ACTIVATION,
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd)
{
	kerd<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_T)>>>(
		ACTIVATION,
		X, Y,
		depart, T,
		x, y,
		p,
		locd);
	ATTENDRE_CUDA();
}

//	============================= Derivation ==============================

static __global__ void kerd_deriv(	//	2 version : 1 stricte et une non stricte
	uint ACTIVATION,
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _y = threadIdx.y + blockIdx.y * blockDim.y;

	if (_t < T && _y < Y) {
		float _locd = locd[(depart+_t)*Y + _y] * dy[(depart+_t)*Y + _y];
		atomicAdd(&dp[_y*(X+1) + (X+1-1)], _locd);
		FOR(0, i, X) {
			atomicAdd(&dx[(depart+_t)*X + i], _locd * p[_y*(X+1) + i]);
			atomicAdd(&dp[_y*(X+1) + i], _locd * x[(depart+_t)*X + i]);
		}
	}
};

void d_nvidia_dot1d_naive(	//	2 versions : 1x stricte et 1x non stricte
	uint ACTIVATION,
	uint X, uint Y,
	uint depart, uint T,
	float * x, float * y,
	float * p,
	float * locd,
	float * dy,
	float * dx,
	float * dp)
{
	kerd_deriv<<<dim3(KERD(T, BLOQUE_T), KERD(Y, BLOQUE_Y)), dim3(BLOQUE_T, BLOQUE_T)>>>(
		ACTIVATION,
		X, Y,
		depart, T,
		x, y,
		p,
		locd,
		dy,
		dx,
		dp);
	ATTENDRE_CUDA();
};