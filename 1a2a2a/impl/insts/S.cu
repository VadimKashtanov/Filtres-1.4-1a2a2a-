#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define EXACTE(assertion) do {			\
	if (assertion) printf("\033[92m");	\
	else printf("\033[91m");			\
} while(0);

PAS_OPTIMISER()
void verifier_S() {
	uint depart = DEPART;
	uint T = 7;
	float  y[depart+T];// = {0.7,0.1,-0.9,0.4};
	y[depart+0]=0.7;
	y[depart+1]=0.1;
	y[depart+2]=-0.9;
	y[depart+3]=+0.4;
	y[depart+4]=-0.4;
	y[depart+5]=-0.1;
	y[depart+6]=+0.5;
	float * y__d = cpu_vers_gpu(y, depart+T);
	float score = (
		SCORE(y[depart+0], prixs[depart+L+0], prixs[depart+0]) + \
		SCORE(y[depart+1], prixs[depart+L+1], prixs[depart+1]) + \
		SCORE(y[depart+2], prixs[depart+L+2], prixs[depart+2]) + \
		SCORE(y[depart+3], prixs[depart+L+3], prixs[depart+3]) + \
		SCORE(y[depart+4], prixs[depart+L+4], prixs[depart+4]) + \
		SCORE(y[depart+5], prixs[depart+L+5], prixs[depart+5]) + \
		SCORE(y[depart+6], prixs[depart+L+6], prixs[depart+6]));// / T;
	float pred = (
		(float)(signe(y[depart+0]) == signe(prixs[depart+L+0]/prixs[depart+0]-1)) + \
		(float)(signe(y[depart+1]) == signe(prixs[depart+L+1]/prixs[depart+1]-1)) + \
		(float)(signe(y[depart+2]) == signe(prixs[depart+L+2]/prixs[depart+2]-1)) + \
		(float)(signe(y[depart+3]) == signe(prixs[depart+L+3]/prixs[depart+3]-1)) + \
		(float)(signe(y[depart+4]) == signe(prixs[depart+L+4]/prixs[depart+4]-1)) + \
		(float)(signe(y[depart+5]) == signe(prixs[depart+L+5]/prixs[depart+5]-1)) + \
		(float)(signe(y[depart+6]) == signe(prixs[depart+L+6]/prixs[depart+6]-1))) / T;
	float dy[depart+T];
	dy[depart+0]=dSCORE(y[depart+0], prixs[depart+L+0], prixs[depart+0]);// / T;
	dy[depart+1]=dSCORE(y[depart+1], prixs[depart+L+1], prixs[depart+1]);// / T; 
	dy[depart+2]=dSCORE(y[depart+2], prixs[depart+L+2], prixs[depart+2]);// / T;
	dy[depart+3]=dSCORE(y[depart+3], prixs[depart+L+3], prixs[depart+3]);// / T;
	dy[depart+4]=dSCORE(y[depart+4], prixs[depart+L+4], prixs[depart+4]);// / T;
	dy[depart+5]=dSCORE(y[depart+5], prixs[depart+L+5], prixs[depart+5]);// / T;
	dy[depart+6]=dSCORE(y[depart+6], prixs[depart+L+6], prixs[depart+6]);// / T;

	titre("Verification SCORES");

	float profondeure = 0.001;

	printf("            manuel  | intel  | nvidia\n");
	float _intel_score = intel_score(y,depart,T);
	float _nvidia_score = nvidia_score(y__d,depart,T);
	EXACTE(fabs(score - _intel_score)<profondeure && fabs(score - _nvidia_score)<profondeure);
	printf("score      %+f %+f %+f\033[0m\n", score, _intel_score, _nvidia_score);
	float _intel_pred = intel_prediction(y,depart,T);
	float _nvidia_pred = nvidia_prediction(y__d,depart,T);
	EXACTE(fabs(pred - _intel_pred)<profondeure && fabs(pred - _nvidia_pred)<profondeure);
	printf("prediction %+f %+f %+f\033[0m\n", pred, _intel_pred, _nvidia_pred);

	printf("------------- dy -----------\n");
	float * _dy = alloc<float>(depart+T);
	float * _dy__d = cudalloc<float>(depart+T);
	d_intel_score(y, _dy, depart, T);
	d_nvidia_score(y__d, _dy__d, depart, T);
	float * _dy__d_cpu = gpu_vers_cpu<float>(_dy__d, depart+T);
	printf("      manuel |  intel   |  nvidia\n");
	FOR(0, i, T) {
		EXACTE(fabs(dy[depart+i]-_dy[depart+i])<profondeure && fabs(dy[depart+i]-_dy__d_cpu[depart+i])<profondeure);
		printf("%2.i| %+f %+f %+f\033[0m\n", i, dy[depart+i], _dy[depart+i], _dy__d_cpu[depart+i]);
	}

	cudafree<float>(_dy__d);
	cudafree<float>(y__d);
	free(_dy);
	free(_dy__d_cpu);
};