#include "hip/hip_runtime.h"
#include "filtres.cuh"

#define BLOQUE_T  4//8
#define BLOQUE_B  4//8
#define BLOQUE_FB 8//16//32//8

#include "../../../impl_tmpl/tmpl_etc.cu"

static __device__ float filtre_device(float * x, float * dif_x, float * f, float * dif_f, float * locd_s, float * locd_d) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	float y = expf(-s*s -d*d);

	locd_s[0] = -2*2*s*y;
	locd_d[0] = -2*2*d*y;

	return 2*y-1;
};

static __global__ void kerd_filtre_naive(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		y[(depart+_t)*(bloques*f_par_bloque) + _b*f_par_bloque + _f] = filtre_device(
			x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			dif_x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			dif_f + _b*f_par_bloque*(N-1) + _f*(N-1),
			locd_s+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			locd_d+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f
		);
	}
};

void nvidia_filtres_naive(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d)
{
	kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		depart, T,
		bloques, f_par_bloque, ligne,
		x, dif_x,
		f, dif_f,
		y,
		locd_s, locd_d);
	ATTENDRE_CUDA();
}

//	======================================================================================================================================

/*static __device__ float d_filtre_device(float * x, float * dif_x, float * f, float * dif_f, float * locd_s, float * locd_d, float * dy, float * df) {
	float ds = locd_s[0] * dy[0] / 8;
	float dd = locd_d[0] * dy[0] / 7;

	atomicAdd(&df[N-1], ds * 1 / (2*sqrtf(1 + fabs(x[N-1] - f[N-1]))) * (-1) * cuda_signe(x[N-1] - f[N-1]));
	FOR(0, i, N-1)
	{
		//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		atomicAdd(&df[i], ds * 1 / (2*sqrtf(1 + fabs(x[i] - f[i]))) * (-1) * cuda_signe(x[i] - f[i]));
		//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
		atomicAdd(&df[i+1], dd * 2 * (1 + fabs(dif_x[i] - dif_f[i])) * cuda_signe(dif_x[i] - dif_f[i]) * (-1));
		atomicAdd(&df[ i ], dd * 2 * (1 + fabs(dif_x[i] - dif_f[i])) * cuda_signe(dif_x[i] - dif_f[i]) * (+1));
	}
};

static __global__ void d_kerd_filtre_naive(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		d_filtre_device(
			x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			dif_x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			f     + _b*f_par_bloque*N      + _f*N,
			dif_f + _b*f_par_bloque*(N-1) + _f*(N-1),
			locd_s+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			locd_d+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			dy + (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			df + _b*f_par_bloque*N + _f*N
		);
	}
};*/

static __global__ void d_kerd_filtre_naive(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	uint thz = threadIdx.z;

	uint thy = threadIdx.y;	// les 8 qui vont atomicAdd les df en chaque point qui lui est associé

	__shared__ float __x[N];
	__shared__ float __diff_x[N-1];
	//__shared__ float __f[N][N];
	//__shared__ float __diff_f[N][N-1];

	FOR(0, ___t, BLOQUE_T) {
		FOR(0, _b, bloques) {
			uint _t = ___t + blockIdx.x * blockDim.x;
			//uint _b = ___b + blockIdx.y * blockDim.y;
			uint _f = threadIdx.z + blockIdx.z * blockDim.z;

			if (_t < T && _b < bloques && _f < f_par_bloque) {
				//
				if (thy == 0) {
					__x[thz] = x[ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR + thz];
					if (thz != N-1)
						__diff_x[thz] = dif_x[ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR + thz];
				}
				//
				//FOR(0, i, N-1) {
				//	__f[thz][i] = f[_b*f_par_bloque*N      + _f*N + thy];
				//if (thy != N-1)
				//	__diff_f[thz][i] = dif_f[_b*f_par_bloque*(N-1) + _f*(N-1) + thy];
				//}
				//__f[thz][N-1] = f[_b*f_par_bloque*N      + _f*N + N-1];
				//
				__syncthreads();
				float * __df = df + _b*f_par_bloque*N + _f*N;
				float * __f = f + _b*f_par_bloque*N + _f*N;
				float * __diff_f = dif_f + _b*f_par_bloque*(N-1) + _f*(N-1);
				//
				float _dy0 = dy[(depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f];
				float ds = locd_s[(depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f] * _dy0 / 8;
				float dd = locd_d[(depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f] * _dy0 / 7;

				/*atomicAdd(&__df[N-1], ds * 1 / (2*sqrtf(1 + fabs(__x[N-1] - __f[thz][N-1]))) * (-1) * cuda_signe(__x[N-1] - __f[thz][N-1]));
				FOR(0, i, N-1)
				{
					//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
					atomicAdd(&__df[i], ds * 1 / (2*sqrtf(1 + fabs(__x[i] - __f[thz][i]))) * (-1) * cuda_signe(x[i] - f[i]));
					//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
					atomicAdd(&__df[i+1], dd * 2 * (1 + fabs(__diff_x[i] - __diff_f[thz][i])) * cuda_signe(__diff_x[i] - __diff_f[thz][i]) * (-1));
					atomicAdd(&__df[ i ], dd * 2 * (1 + fabs(__diff_x[i] - __diff_f[thz][i])) * cuda_signe(__diff_x[i] - __diff_f[thz][i]) * (+1));
				}*/
				//atomicAdd(&__df[N-1], ds * 1 / (2*sqrtf(1 + fabs(__x[N-1] - __f[thz][N-1]))) * (-1) * cuda_signe(__x[N-1] - __f[thz][N-1]));
				atomicAdd(&__df[thy], ds * 1 / (2*sqrtf(1 + fabs(__x[thy] - __f[thy]))) * (-1) * cuda_signe(x[thy] - __f[thy]));
				if (thy != N-1) {
					atomicAdd(&__df[thy+1], dd * 2 * (1 + fabs(__diff_x[thy] - __diff_f[thy])) * cuda_signe(__diff_x[thy] - __diff_f[thy]) * (-1));
					atomicAdd(&__df[ thy ], dd * 2 * (1 + fabs(__diff_x[thy] - __diff_f[thy])) * cuda_signe(__diff_x[thy] - __diff_f[thy]) * (+1));
				}
				__syncthreads();
			}
		}
	}
};

void d_nvidia_filtres_naive(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	d_kerd_filtre_naive<<<dim3(DIV(T, BLOQUE_T), 1, KERD(f_par_bloque, BLOQUE_FB)), dim3(1, N, BLOQUE_FB)>>>(
		depart, T,
		bloques, f_par_bloque, ligne,
		x, dif_x,
		f, dif_f,
		y,
		locd_s, locd_d,
		dy,
		df);
	ATTENDRE_CUDA();
}

/*void d_nvidia_filtres_naive(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	d_kerd_filtre_naive<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		depart, T,
		bloques, f_par_bloque, ligne,
		x, dif_x,
		f, dif_f,
		y,
		locd_s, locd_d,
		dy,
		df);
	ATTENDRE_CUDA();
}*/