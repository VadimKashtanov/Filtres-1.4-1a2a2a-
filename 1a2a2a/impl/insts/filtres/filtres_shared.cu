#include "hip/hip_runtime.h"
#include "filtres.cuh"

#define BLOQUE_T  4
#define BLOQUE_B  4
#define BLOQUE_FB N

static __device__ float filtre_device_shared(float * x, float * dif_x, float * f, float * dif_f, float * locd_s, float * locd_d) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	float y = 2*expf(-s*s -d*d)-1;

	locd_s[0] = -4*y*s;
	locd_d[0] = -4*y*d;

	return y;
};

static __global__ void kerd_filtre_shared(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

	if (_t < T && _b < bloques && _f < f_par_bloque) {
		y[(depart+_t)*(bloques*f_par_bloque) + _b*f_par_bloque + _f] = filtre_device_shared(
			x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			dif_x + ligne[_b]*PRIXS*N_FLTR + (depart+_t)*N_FLTR,
			f     + _b*f_par_bloque*N     + _f*N,
			dif_f + _b*f_par_bloque*(N-1) + _f*(N-1),
			locd_s+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f,
			locd_d+ (depart+_t)*bloques*f_par_bloque + _b*f_par_bloque + _f
		);
	}
};

/*
__global__ static
void mdt1_filtre__kerd(
	uint t0,
	uint * EMA_INT_BLOQUE_ligne__d,
	float * normalisee__d,
	float * f, float * dif_f,
	float * y, uint Y_MAX)
{
	//	<<<(T, BLOQUES), (BLOQUE_ST)>>> //BLOQUE_ST*N_FLT==256

	uint _t = threadIdx.x + blockIdx.x * blockDim.x;
	uint _b = threadIdx.y + blockIdx.y * blockDim.y;
	uint _f = threadIdx.z + blockIdx.z * blockDim.z;

#define bloque blockIdx.y
#define f_dans_bloque threadIdx.x

	uint t = blockIdx.x;

	//	__partage__
	__shared__ uint ligne;
	if (threadIdx.x == 0)
		ligne = EMA_INT_BLOQUE_ligne__d[bloque];

	//	__partage__
	__shared__ float sh_x[N_FLTR];
	if (f_dans_bloque < N_FLTR)
		sh_x[f_dans_bloque] = normalisee__d[ligne*PRIXS*N_FLTR + (t0+t)*N_FLTR + f_dans_bloque];

	//	fonction de filtre
	float s = 0, d = 0;
	FOR(0, i, N_FLTR-1) {
		s += sqrtf(1 + fabs(     sh_x[i]        -   f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + i]  ));
		d += powf((1 + fabs((sh_x[i+1]-sh_x[i]) - dif_f[bloque*BLOQUE_ST*(N_FLTR-1) + f_dans_bloque*(N_FLTR-1) + i])), 2);
	};
	s += sqrtf(1 + fabs(sh_x[N_FLTR-1] - f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + N_FLTR-1]));

	//	s = a + b + c ...

	s = s/8-1;
	d = d/7-1;

	y[(depart+_t)*(bloques*f_par_bloque) + _b*f_par_bloque + _f] = 2*expf(-s*s -d*d)-1;
};*/

void nvidia_filtres_shared(	//	2 version : 1 stricte et une non stricte
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d)
{
	kerd_filtre_shared<<<dim3(KERD(T, BLOQUE_T), KERD(bloques, BLOQUE_B), KERD(f_par_bloque, BLOQUE_FB)), dim3(BLOQUE_T, BLOQUE_B, BLOQUE_FB)>>>(
		depart, T,
		bloques, f_par_bloque, ligne,
		x, dif_x,
		f, dif_f,
		y,
		locd_s, locd_d);
	ATTENDRE_CUDA();
};

//	=================================================

void d_nvidia_filtres_shared(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	TODO()
}