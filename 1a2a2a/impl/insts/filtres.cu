#include "hip/hip_runtime.h"
#include "filtres.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

//
//	//
//	//	//	2*y-1  ===> y tout court
//	//
//

static float filtre(float * x, float * dif_x, float * f, float * dif_f, float * locd_s, float * locd_d) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	float y = expf(-s*s -d*d);

	locd_s[0] = -2*2*s*y;
	locd_d[0] = -2*2*d*y;

	return 2*y-1;
};

void intel_filtres(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d)
{
//#pragma omp parallel
//#pragma omp for
	FOR(0, t, T) {
		FOR(0, b, bloques) {
			FOR(0, _f, f_par_bloque) {
				y[(depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f] = filtre(
						x + ligne[b]*PRIXS*N_FLTR + (depart+t)*N_FLTR,
					dif_x + ligne[b]*PRIXS*N_FLTR + (depart+t)*N_FLTR,
					f     + b*f_par_bloque*N     + _f*N,
					dif_f + b*f_par_bloque*(N-1) + _f*(N-1),
					locd_s+ (depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f,
					locd_d+ (depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f
				);
			}
		}
	}
}

static void d_filtre(float * x, float * dif_x, float * f, float * dif_f, float * locd_s, float * locd_d, float * dy, float * df) {
	float ds = locd_s[0] * dy[0] / 8;
	float dd = locd_d[0] * dy[0] / 7;

	df[N-1] += ds * 1 / (2*sqrtf(1 + fabs(x[N-1] - f[N-1]))) * (-1) * signe(x[N-1] - f[N-1]);
	FOR(0, i, N-1)
	{
		//s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		df[i] += ds * 1 / (2*sqrtf(1 + fabs(x[i] - f[i]))) * (-1) * signe(x[i] - f[i]);
		//d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
		df[i+1] += dd * 2 * (1 + fabs(dif_x[i] - dif_f[i])) * signe(dif_x[i] - dif_f[i]) * (-1);
		df[ i ] += dd * 2 * (1 + fabs(dif_x[i] - dif_f[i])) * signe(dif_x[i] - dif_f[i]) * (+1);
	}
};

void  d_intel_filtres(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df)
{
	FOR(0, t, T) {
		FOR(0, b, bloques) {
			FOR(0, _f, f_par_bloque) {
				d_filtre(
						x + ligne[b]*PRIXS*N_FLTR + (depart+t)*N_FLTR,
					dif_x + ligne[b]*PRIXS*N_FLTR + (depart+t)*N_FLTR,
					f     + b*f_par_bloque*N     + _f*N,
					dif_f + b*f_par_bloque*(N-1) + _f*(N-1),
					locd_s+ (depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f,
					locd_d+ (depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f,
					dy + (depart+t)*bloques*f_par_bloque + b*f_par_bloque + _f,
					df     + b*f_par_bloque*N     + _f*N
				);
			}
		}
	}
}

void nvidia_filtres(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	uint mode)
{
	if (mode == 0) {
		nvidia_filtres_naive(
			depart, T,
			bloques, f_par_bloque, ligne,
			x, dif_x,
			f, dif_f,
			y,
			locd_s, locd_d);
	} else if (mode == 1) {
		nvidia_filtres_shared(
			depart, T,
			bloques, f_par_bloque, ligne,
			x, dif_x,
			f, dif_f,
			y,
			locd_s, locd_d);
	} else {
		ERR("Pas de mode %i pour mes filtres", mode);
	}
};

void d_nvidia_filtres(
	uint depart, uint T,
	uint bloques, uint f_par_bloque, uint * ligne,
	float * x, float * dif_x,
	float * f, float * dif_f,
	float * y,
	float * locd_s, float * locd_d,
	float * dy,
	float * df,
	uint mode)
{
	if (mode == 0) {
		d_nvidia_filtres_naive(
			depart, T,
			bloques, f_par_bloque, ligne,
			x, dif_x,
			f, dif_f,
			y,
			locd_s, locd_d,
			dy,
			df);
	} else if (mode == 1) {
		d_nvidia_filtres_shared(
			depart, T,
			bloques, f_par_bloque, ligne,
			x, dif_x,
			f, dif_f,
			y,
			locd_s, locd_d,
			dy,
			df);
	} else {
		ERR("Pas de mode %i pour mes filtres", mode);
	}
};

static void verifier_nvidia_intel() {
	uint depart = DEPART;
	uint T = 2*16;
	uint bloques = 16;
	uint f_par_bloque = 16;

	uint ligne[bloques] = {1,7,6,8,4,3,9,4,5,6,7,4,5,1,2,3};
	float * f = lst_rnd(bloques*f_par_bloque*N, -1, 1);
	float * diff_f = alloc<float>(bloques*f_par_bloque*(N-1));
	FOR(0, b, bloques*f_par_bloque) {
		FOR(0, i, N-1) {
			diff_f[b*(N-1)+i] = f[b*N + i+1] - f[b*N + i];
		}
	}
	float * y = alloc<float>((depart+T)*f_par_bloque*bloques);

	uint * ligne__d = cpu_vers_gpu<uint>(ligne, bloques);
	float * f__d = cpu_vers_gpu<float>(f, bloques*f_par_bloque*N);
	float * diff_f__d = cpu_vers_gpu<float>(diff_f, bloques*f_par_bloque*(N-1));
	float * y__d = cudalloc<float>((depart+T)*f_par_bloque*bloques);

	float * locd_s = alloc<float>(bloques*f_par_bloque*(DEPART+T));
	float * locd_d = alloc<float>(bloques*f_par_bloque*(DEPART+T));

	float * locd_s__d = cudalloc<float>(bloques*f_par_bloque*(DEPART+T));
	float * locd_d__d = cudalloc<float>(bloques*f_par_bloque*(DEPART+T));

	intel_filtres(
		depart, T,
		bloques, f_par_bloque, ligne,
		normalisee, dif_normalisee,
		f, diff_f,
		y,
		locd_s, locd_d);

	FOR(0, mode, 2) {
		CONTROLE_CUDA(hipMemset(y__d, 0, sizeof(float) * T*f_par_bloque*bloques));
		//
		nvidia_filtres(
			depart, T,
			bloques, f_par_bloque, ligne__d,
			normalisee__d, dif_normalisee__d,
			f__d, diff_f__d,
			y__d,
			locd_s__d, locd_d__d,
			mode
		);
		printf("########## Verif filtre MODE %i #######\n", mode);

		puts("y__d\n");
		comparer_lst(
			y+depart*f_par_bloque*bloques,
			gpu_vers_cpu<float>(y__d, (depart+T)*f_par_bloque*bloques) + depart*f_par_bloque*bloques,
			T*f_par_bloque*bloques,
			0.001
		);
	}
};

void verif_filtres() {
	MSG("Shared filtres pas encore fait. Juste un copier-coller de la version naive");
	verifier_nvidia_intel();
};