#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

float pourcent_masque_nulle[C] = {
	0,
	0,
	0,0,
	0,0,0,
	0,0,0,0,
	0,0,0,
	0,0,0,0,
	0,0,0,0,
	0
};

float pourcent_masque[C] = {
	0.,
	0.20,
	0.10,0.10,
	0.10,0.10,0.10,
	0.10,0.10,0.10,0.10,
	0.10,0.10,0.10,
	0.0,0.0,0.0,0.0,
	0,0,0,0,
	0
};

float * alpha = de_a(1e-3, 1e-3, C);

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	//	-- Init --
	srand(0);
	hipSetDevice(0);

	titre(" Charger tout ");  charger_tout();
	//titre("Verifier  FILTRES"); verif_filtres();
	//titre("Verifier  DOT1D"); 	 verif_do1d();
	//titre("Verifier  S"); 		 verifier_S();
	//titre("Verifier MDL");      verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");

	/*uint ST[C] = {
		512,
		256,
		128, 128,
		64, 64, 64,
		32, 32, 32, 32,
		16, 16, 16,
		8, 8, 8, 8,
		4, 4, 4, 4,
		1
	};
	uint activations[C] = {TANH};
	uint bloques      = 64;
	uint f_par_bloque =  8;
	uint lignes[bloques] = {
		0,0,0,0,0,0,0,0,0,0,0,0,
	    1,1,1,1,1,
	    2,2,2,2,2,
	    3,3,3,3,
	    4,4,4,4,4,
	    5,5,5,6,6,7,7,7,7,8,9,9,9,9,10,11,12,13,14,
	    18,19,20,21,21,21,22,23,23,24,25,
	    28,
	    31,31
	};
	Mdl_t * mdl = cree_mdl(ST, activations, bloques, f_par_bloque, lignes);*/

	Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, 16);
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%32=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%16);
	//
	float ancien = 100*mdl_pred(mdl, t0, t1, 3);
	printf("%%%% pred = %f\n", ancien);
	//
	INIT_CHRONO(chrono)
	//
	DEPART_CHRONO(chrono)
	mdl_aller_retour(mdl, t0, t1, 3);
	float sec_opti = VALEUR_CHRONO(chrono);
	uint OPTIMISATIONS = 150*1500;
	printf("\033[3;92;m1 opti = %+f s, donc %i*%+f = %+f s = %+f mins\033[0m\n",
		sec_opti,
		OPTIMISATIONS, sec_opti,
		OPTIMISATIONS * sec_opti,
		OPTIMISATIONS * sec_opti / 60.0);
	//
	uint REP = 150;
	FOR(0, rep, REP) {
		FOR(0, i, 1) {
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*200,
				alpha, 1.0,
				RMSPROP, 1500,
				pourcent_masque);
			printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}
		/*FOR(0, i, 5) {
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*800,
				alpha, 1.0,
				RMSPROP, 1500,
				//pourcent_masque_simple);
				pourcent_masque);
			printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}*/
		//
		/*optimiser(
			mdl,
			t0, t1,
			alpha, 1.0,
			RMSPROP, 2000,
			pourcent_masque_nulle);*/
		//
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, REP);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};