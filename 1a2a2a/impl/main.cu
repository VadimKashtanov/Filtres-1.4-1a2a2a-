#include "main.cuh"

#include "../impl_tmpl/tmpl_etc.cu"

float pourcent_masque_nulle[C] = {0};

float * alpha_moyen__paquets16X500 = de_a(5e-4, 4e-5, C);
float * alpha_initialisation = de_a(1e-3, 1e-4, C);

float alpha_zeropoids[C] = {
	5e-3,
	0,0,0,0,0,0,0,0,0, 0, 0, 0, 0, 0, 0
};

float * alpha_initialisation__PRIXS = de_a(1e-3, 1e-4, C);

float alpha_zeropoids__PRIXS[C] = {
	3e-2,
	0,0,0,0,0,0,0,0,0, 0, 0, 0, 0, 0, 0
};


PAS_OPTIMISER()
int main(int argc, char ** argv) {
	//	-- Init --
	srand(0);
	hipSetDevice(0);

	titre(" Charger tout ");  charger_tout();
	//titre("Verifier  FILTRES"); verif_filtres();
	//titre("Verifier  DOT1D"); 	 verif_do1d();
	//titre("Verifier  S"); 		 verifier_S();
	//titre("Verifier MDL");      verif_mdl_1e5();

	//===============
	titre("  Programme Generale  ");

	uint ST[C] = {
		512,
		256, 256,
		128, 128,
		64, 64, 64, 64, 64, 64,
		32, 32,
		16, 16,
		8, 8,
		4, 4,
		1
	};
	uint bloques      = 64;
	uint f_par_bloque =  8;
	uint lignes[bloques] = {
	};
	Mdl_t * mdl = cree_mdl(ST, bloques, f_par_bloque, lignes);

	//Mdl_t * mdl = ouvrire_mdl("mdl.bin");

	plumer_mdl(mdl);

	//	================= Initialisation ==============
	uint t0 = DEPART;
	uint t1 = ROND_MODULO(FIN, 16);
	printf("t0=%i t1=%i FIN=%i (t1-t0=%i, %%32=%i)\n", t0, t1, FIN, t1-t0, (t1-t0)%16);
	//
	float ancien = 100*mdl_pred(mdl, t0, t1, 3);
	printf("%%%% pred = %f\n", ancien);
	//
	INIT_CHRONO(chrono)
	//
	DEPART_CHRONO(chrono)
	mdl_aller_retour(mdl, t0, t1, 3);
	float sec_opti = VALEUR_CHRONO(chrono);
	uint OPTIMISATIONS = 10*100;
	printf("\033[3;92;m1 opti = %+f s, donc %i*%+f = %+f s = %+f mins\033[0m\n",
		sec_opti,
		OPTIMISATIONS, sec_opti,
		OPTIMISATIONS * sec_opti,
		OPTIMISATIONS * sec_opti / 60.0);
	//mdl_plume_grad(mdl, t0, t1);
	//
	FOR(0, rep, 100) {
		/*FOR(0, i, 3) {
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*400,
				alpha_initialisation, 1.0,
				RMSPROP, 1000,
				//pourcent_masque_simple);
				pourcent_masque_nulle);
			printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}
		FOR(0, i, 2) {
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*200,
				alpha_zeropoids, 1.0,
				RMSPROP, 200,
				//pourcent_masque_simple);
				pourcent_masque_nulle);
			printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}*/
		/*FOR(0, i, 10) {
			optimisation_mini_packet(
				mdl,
				t0, t1, 16*40,
				alpha_zeropoids, 1.0,
				RMSPROP, 30,
				//pourcent_masque_simple);
				pourcent_masque_nulle);
			printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
			mdl_gpu_vers_cpu(mdl);
			ecrire_mdl(mdl, "mdl.bin");
		}
		*/
		//
		optimiser(
			mdl,
			t0, t1,
			alpha_initialisation__PRIXS, 1.0,
			SGD, 300,
			pourcent_masque_nulle);
		/*optimiser(
			mdl,
			t0, t1,
			alpha_zeropoids__PRIXS, 1.0,
			RMSPROP, 100,
			pourcent_masque_nulle);*/
		//
		mdl_gpu_vers_cpu(mdl);
		ecrire_mdl(mdl, "mdl.bin");
		printf("SCORE GENERALE : pred = %f\n", 100*mdl_pred(mdl, t0, t1, 3));
		printf("===================================================\n");
		printf("==================TERMINE %i/%i=======================\n", rep+1, 100);
		printf("===================================================\n");
	}
	//
	mdl_gpu_vers_cpu(mdl);
	ecrire_mdl(mdl, "mdl.bin");
	liberer_mdl(mdl);

	//	-- Fin --
	liberer_tout();
};